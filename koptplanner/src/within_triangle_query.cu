#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#define det(u_x, u_y, v_x, v_y) (u_x*v_y-u_y*v_x)


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * Checks whether the point tested lies within the triangles passed.
 * Algorithm according to https://mathworld.wolfram.com/TriangleInterior.html
 * */
__global__ void within_triangle_query(float *p_x,
                                        float *p_y,
                                        float *v0_x, 
                                        float *v0_y,
                                        float *v1_x, 
                                        float *v1_y, 
                                        float *v2_x, 
                                        float *v2_y, 
                                        bool *output)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float d1_x = v1_x[i] - v0_x[i];
    float d1_y = v1_y[i] - v0_y[i];
    float d2_x = v2_x[i] - v0_x[i];
    float d2_y = v2_y[i] - v0_y[i];
    
    float denominator = det(d1_x,d1_y,d2_x,d2_y);
    float a = (det(p_x[0],p_y[0],d2_x,d2_y) - det(v0_x[i],v0_y[i],d2_x,d2_y)) / denominator;
    float b = - (det(p_x[0],p_y[0],d1_x,d1_y) - det(v0_x[i],v0_y[i],d1_x,d1_y)) / denominator;
    
    if(a>0 && b>0 && a+b<1) output[i] = true;
    else output[i] = false;
}

/**
 * CPU-Version of CUDA-Kernel-Code to assess performance
 * */
void within_triangle_query_cpu(float p_x,
    float p_y,
    float *v0_x, 
    float *v0_y,
    float *v1_x, 
    float *v1_y, 
    float *v2_x, 
    float *v2_y, 
    bool *output,
    int size)
{
    for (int i=0; i<size; i++)
    {
        float d1_x = v1_x[i] - v0_x[i];
        float d1_y = v1_y[i] - v0_y[i];
        float d2_x = v2_x[i] - v0_x[i];
        float d2_y = v2_y[i] - v0_y[i];

        float denominator = det(d1_x,d1_y,d2_x,d2_y);
        float a = (det(p_x,p_y,d2_x,d2_y) - det(v0_x[i],v0_y[i],d2_x,d2_y)) / denominator;
        float b = - (det(p_x,p_y,d1_x,d1_y) - det(v0_x[i],v0_y[i],d1_x,d1_y)) / denominator;

        if(a>0 && b>0 && a+b<1) output[i] = true;
        else output[i] = false;
    }
}

/**
 * Allocates GPU-Memory for input arrays and copies their contents to the device. Launches CUDA-Kernel 
 * that determines whether the tested points lie within the triangle or not. Results are copied back into 
 * the output array
 * \param num Size of input arrays
 * \param threads Threads per block parameter for CUDA-Kernel launch
 * \param p Coordinates of point being tested for occlusion
 * \param v Arrays of x and y coordinates of the three vertices defining each triangle
 * \param output Array in which the computation results are written. Boolean array of size num must be allocated. 
 * \param use_gpu Specify whether the CUDA-GPU-API or a fallback CPU implementation is to be used
 * */
int *within_triangle_query_gpu_driver(int num, int threads ,
    float p_x,
    float p_y,
    float *v0_x, 
    float *v0_y,
    float *v1_x, 
    float *v1_y, 
    float *v2_x, 
    float *v2_y, 
    bool *output,
    bool use_gpu) 
{
    if(use_gpu)
    {
        // device copies of inputs and output
        float *d_p_x, *d_p_y;
        float *d_v0_x, *d_v0_y, *d_v1_x, *d_v1_y, *d_v2_x, *d_v2_y;
        bool *d_output;
        int size_float_arr = num*sizeof(float);
        int size_bool_arr = num*sizeof(bool);

        // Alloc space for device copies of vectors
        gpuErrchk  ( hipMalloc((void **)&d_p_x, sizeof(float)));//Only a single number, not an array
        gpuErrchk  ( hipMalloc((void **)&d_p_y, sizeof(float)));
        gpuErrchk  ( hipMalloc((void **)&d_v0_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v0_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v1_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v1_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v2_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v2_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_output, size_bool_arr));

        // Copy inputs to device
        gpuErrchk  ( hipMemcpy(d_p_x, &p_x, sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_p_y, &p_y, sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v0_x, v0_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v0_y, v0_y, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v1_x, v1_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v1_y, v1_y, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v2_x, v2_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v2_y, v2_y, size_float_arr, hipMemcpyHostToDevice));

        // Launch kernel on GPU
        //TO-DO: Determine optimum value for number of blocks
        within_triangle_query<<<126, threads>>>
            (d_p_x,
            d_p_y,
            d_v0_x, 
            d_v0_y,
            d_v1_x, 
            d_v1_y, 
            d_v2_x, 
            d_v2_y, 
            d_output);

        gpuErrchk( hipPeekAtLastError() );
        // Wait for the GPU to finish
        gpuErrchk  ( hipDeviceSynchronize());
        // Copy result back to host
        gpuErrchk  ( hipMemcpy(output, d_output, size_bool_arr, hipMemcpyDeviceToHost));

        // Cleanup
        hipFree(d_p_x);
        hipFree(d_p_y);
        hipFree(d_v0_x );
        hipFree(d_v0_y);
        hipFree(d_v1_x );
        hipFree(d_v1_y );
        hipFree(d_v2_x );
        hipFree(d_v2_y );
        hipFree(d_output);

        // //Check against CPU version
        // bool *output_cpu;
        // output_cpu = (bool*)malloc(num*sizeof(bool));
        // within_triangle_query_cpu(
        //     p_x,
        //     p_y,
        //     v0_x, 
        //     v0_y,
        //     v1_x, 
        //     v1_y, 
        //     v2_x, 
        //     v2_y, 
        //     output_cpu,
        //     num);

        // for(int i=0; i<num; i++)
        // {
        //     if(output[i] != output_cpu[i])
        //     {
        //         ROS_INFO("False GPU-Result at number %i", i);
        //     }
        // }
        // free(output_cpu);
    }
    else
    {
        //CPU-Version only
        within_triangle_query_cpu(
            p_x,
            p_y,
            v0_x, 
            v0_y,
            v1_x, 
            v1_y, 
            v2_x, 
            v2_y, 
            output,
            num);
    }
    
    return 0;
}
