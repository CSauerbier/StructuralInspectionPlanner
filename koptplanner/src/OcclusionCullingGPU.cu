#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <vector>
#include "Culling/Coordinates.hpp"
#include <math.h>
#include <stdexcept>

#define det(u_x, u_y, v_x, v_y) (u_x*v_y-u_y*v_x)

__device__ const float EPSILON = 0.000001;
__device__ const float TOLERANCE = 0.001;    //TO-DO


bool g_use_gpu;
bool g_is_initialized = false;

const unsigned int THREADS_PER_BLOCK = 512;
unsigned int GRID_SIZE;
unsigned int KERNEL_COUNT;

float3 *d_viewpoints;
float3 *d_test_points;
float3 *d_vertex0, *d_vertex1, *d_vertex2;
bool *d_outputs;
int g_vp_size;
int g_mesh_size;
int g_vertices_size;

float3 *h_viewpoints;
float3 *h_test_points;
float3 *h_vertex0, *h_vertex1, *h_vertex2;
bool *h_outputs;

//Cuda Error Handler
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Overloading operators and defining mathematical functions for CUDA float3
//Vector addition
__host__ __device__ float3 operator+(const float3 &a, const float3 &b) 
{
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

//Vector subtraction
__host__ __device__ float3 operator-(const float3 &a, const float3 &b) 
{
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

//Scalar-Vector multiplication
__host__ __device__ float3 operator*(const float &a, const float3 &b)
{
    return make_float3(a*b.x, a*b.y, a*b.z);
}

//Dot product
__host__ __device__ float dot(const float3 &a, const float3 &b) 
{
    return (a.x*b.x + a.y*b.y + a.z*b.z);
}

//Cross product
__host__ __device__ float3 cross(const float3 &a, const float3 &b) 
{
    return make_float3( a.y*b.z - a.z*b.y, 
                        a.z*b.x - a.x*b.z, 
                        a.x*b.y - a.y*b.x);
}

//Euclidian vector norm (Length of vector)
__host__ __device__ float norm(const float3 &a)
{
    return sqrt(dot(a,a));
}


/**
 * Checks whether the point tested lies within the triangles passed.
 * Algorithm according to https://mathworld.wolfram.com/TriangleInterior.html
 * */
__global__ void within_triangle_query(float *p_x,
                                        float *p_y,
                                        float *v0_x, 
                                        float *v0_y,
                                        float *v1_x, 
                                        float *v1_y, 
                                        float *v2_x, 
                                        float *v2_y, 
                                        bool *output)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float d1_x = v1_x[i] - v0_x[i];
    float d1_y = v1_y[i] - v0_y[i];
    float d2_x = v2_x[i] - v0_x[i];
    float d2_y = v2_y[i] - v0_y[i];
    
    float denominator = det(d1_x,d1_y,d2_x,d2_y);
    float a = (det(p_x[0],p_y[0],d2_x,d2_y) - det(v0_x[i],v0_y[i],d2_x,d2_y)) / denominator;
    float b = - (det(p_x[0],p_y[0],d1_x,d1_y) - det(v0_x[i],v0_y[i],d1_x,d1_y)) / denominator;
    
    if(a>0 && b>0 && a+b<1) output[i] = true;
    else output[i] = false;
}

/**
 * CPU-Version of CUDA-Kernel-Code to assess performance
 * */
void within_triangle_query_cpu(float p_x,
    float p_y,
    float *v0_x, 
    float *v0_y,
    float *v1_x, 
    float *v1_y, 
    float *v2_x, 
    float *v2_y, 
    bool *output,
    int size)
{
    for (int i=0; i<size; i++)
    {
        float d1_x = v1_x[i] - v0_x[i];
        float d1_y = v1_y[i] - v0_y[i];
        float d2_x = v2_x[i] - v0_x[i];
        float d2_y = v2_y[i] - v0_y[i];

        float denominator = det(d1_x,d1_y,d2_x,d2_y);
        float a = (det(p_x,p_y,d2_x,d2_y) - det(v0_x[i],v0_y[i],d2_x,d2_y)) / denominator;
        float b = - (det(p_x,p_y,d1_x,d1_y) - det(v0_x[i],v0_y[i],d1_x,d1_y)) / denominator;

        if(a>0 && b>0 && a+b<1) output[i] = true;
        else output[i] = false;
    }
}

/**
 * Allocates GPU-Memory for input arrays and copies their contents to the device. Launches CUDA-Kernel 
 * that determines whether the tested points lie within the triangle or not. Results are copied back into 
 * the output array
 * \param num Size of input arrays
 * \param threads Threads per block parameter for CUDA-Kernel launch
 * \param p Coordinates of point being tested for occlusion
 * \param v Arrays of x and y coordinates of the three vertices defining each triangle
 * \param output Array in which the computation results are written. Boolean array of size num must be allocated. 
 * \param use_gpu Specify whether the CUDA-GPU-API or a fallback CPU implementation is to be used
 * */
int *within_triangle_query_gpu_driver(int num,
    float p_x,
    float p_y,
    float *v0_x, 
    float *v0_y,
    float *v1_x, 
    float *v1_y, 
    float *v2_x, 
    float *v2_y, 
    bool *output,
    bool g_use_gpu) 
{
    if(g_use_gpu)
    {
        // device copies of inputs and output
        float *d_p_x, *d_p_y;
        float *d_v0_x, *d_v0_y, *d_v1_x, *d_v1_y, *d_v2_x, *d_v2_y;
        bool *d_output;
        int size_float_arr = num*sizeof(float);
        int size_bool_arr = num*sizeof(bool);

        // Alloc space for device copies of vectors
        gpuErrchk  ( hipMalloc((void **)&d_p_x, sizeof(float)));//Only a single number, not an array
        gpuErrchk  ( hipMalloc((void **)&d_p_y, sizeof(float)));
        gpuErrchk  ( hipMalloc((void **)&d_v0_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v0_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v1_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v1_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v2_x, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_v2_y, size_float_arr));
        gpuErrchk  ( hipMalloc((void **)&d_output, size_bool_arr));

        // Copy inputs to device
        gpuErrchk  ( hipMemcpy(d_p_x, &p_x, sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_p_y, &p_y, sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v0_x, v0_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v0_y, v0_y, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v1_x, v1_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v1_y, v1_y, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v2_x, v2_x, size_float_arr, hipMemcpyHostToDevice));
        gpuErrchk  ( hipMemcpy(d_v2_y, v2_y, size_float_arr, hipMemcpyHostToDevice));

        // Launch kernel on GPU
        //TO-DO: Determine optimum value for launch parameters
        within_triangle_query<<<num, THREADS_PER_BLOCK>>>
            (d_p_x,
            d_p_y,
            d_v0_x, 
            d_v0_y,
            d_v1_x, 
            d_v1_y, 
            d_v2_x, 
            d_v2_y, 
            d_output);

        gpuErrchk( hipPeekAtLastError() );
        // Wait for the GPU to finish
        gpuErrchk  ( hipDeviceSynchronize());
        // Copy result back to host
        gpuErrchk  ( hipMemcpy(output, d_output, size_bool_arr, hipMemcpyDeviceToHost));

        // Cleanup
        hipFree(d_p_x);
        hipFree(d_p_y);
        hipFree(d_v0_x );
        hipFree(d_v0_y);
        hipFree(d_v1_x );
        hipFree(d_v1_y );
        hipFree(d_v2_x );
        hipFree(d_v2_y );
        hipFree(d_output);
    }
    else
    {
        //CPU-Version only
        within_triangle_query_cpu(
            p_x,
            p_y,
            v0_x, 
            v0_y,
            v1_x, 
            v1_y, 
            v2_x, 
            v2_y, 
            output,
            num);
    }
    
    return 0;
}

/**
 * Writes data to both machine RAM and GPU RAM as c-style arrays. Sets size variables according to the input data
 * \param tri_v Vector of mesh triangles
 * \param test_points Vector of points to be tested for occlusion
 * \param view_points Camera positions to be tested
 * \param use_gpu Specify whether to execute calculations on the GPU or to fall back to CPU execution 
 */
void setGeometryData_interface(std::vector<TriangleVertices*> &tri_v, std::vector<CartesianCoordinates*> &test_points, std::vector<CartesianCoordinates*> view_points, bool use_gpu)
{
    /*Mesh------------------*/
    g_use_gpu = use_gpu;
    g_mesh_size = tri_v.size();
    g_vertices_size = test_points.size();

    GRID_SIZE = (g_mesh_size+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
    KERNEL_COUNT = GRID_SIZE*THREADS_PER_BLOCK;

    if(GRID_SIZE == 0 || THREADS_PER_BLOCK == 0)
    {
        throw std::runtime_error("CUDA launch parameters cannot be zero");
    }

    h_vertex0 = (float3*)malloc(KERNEL_COUNT*sizeof(float3));
    h_vertex1 = (float3*)malloc(KERNEL_COUNT*sizeof(float3));
    h_vertex2 = (float3*)malloc(KERNEL_COUNT*sizeof(float3));
    h_test_points = (float3*)malloc(g_vertices_size*sizeof(float3));
    h_outputs = (bool*)malloc(g_vertices_size*sizeof(bool));


    //Fill c array with mesh data passed in
    for(int i=0; i<g_mesh_size; i++)
    {
        h_vertex0[i] = make_float3(tri_v[i]->vertices[0]->x, tri_v[i]->vertices[0]->y, tri_v[i]->vertices[0]->z);
        h_vertex1[i] = make_float3(tri_v[i]->vertices[1]->x, tri_v[i]->vertices[1]->y, tri_v[i]->vertices[1]->z);
        h_vertex2[i] = make_float3(tri_v[i]->vertices[2]->x, tri_v[i]->vertices[2]->y, tri_v[i]->vertices[2]->z);
    }
    //Fill remaining array values with zeroes
    for (int i=g_mesh_size; i<KERNEL_COUNT; i++)
    {
        h_vertex0[i] = make_float3(0,0,0);
        h_vertex1[i] = make_float3(0,0,0);
        h_vertex2[i] = make_float3(0,0,0);
    }
    for(int i=0; i<g_vertices_size; i++)
    {
        h_test_points[i] = make_float3(test_points[i]->x, test_points[i]->y, test_points[i]->z);
        h_outputs[i] = true;
    }

    //Allocate space on GPU
    gpuErrchk( hipMalloc((void **)&d_vertex0, KERNEL_COUNT*sizeof(float3)));
    gpuErrchk( hipMalloc((void **)&d_vertex1, KERNEL_COUNT*sizeof(float3)));
    gpuErrchk( hipMalloc((void **)&d_vertex2, KERNEL_COUNT*sizeof(float3)));
    gpuErrchk( hipMalloc((void **)&d_test_points, g_vertices_size*sizeof(float3)));
    gpuErrchk( hipMalloc((void **)&d_outputs, g_vertices_size*sizeof(bool)));

    //Copy to device
    gpuErrchk  ( hipMemcpy(d_vertex0, h_vertex0, KERNEL_COUNT*sizeof(float3), hipMemcpyHostToDevice));
    gpuErrchk  ( hipMemcpy(d_vertex1, h_vertex1, KERNEL_COUNT*sizeof(float3), hipMemcpyHostToDevice));
    gpuErrchk  ( hipMemcpy(d_vertex2, h_vertex2, KERNEL_COUNT*sizeof(float3), hipMemcpyHostToDevice));
    gpuErrchk  ( hipMemcpy(d_test_points, h_test_points, g_vertices_size*sizeof(float3), hipMemcpyHostToDevice));
    gpuErrchk  ( hipMemcpy(d_outputs, h_outputs, g_vertices_size*sizeof(bool), hipMemcpyHostToDevice));

    /*View-Points--------------*/
    g_vp_size = view_points.size();

    h_viewpoints = (float3*)malloc(g_vp_size*sizeof(float3));

    for (int i=0; i<g_vp_size; i++)
    {
        h_viewpoints[i] = make_float3(view_points[i]->x, view_points[i]->y, view_points[i]->z);
    }

    gpuErrchk( hipMalloc((void **)&d_viewpoints, g_vp_size*sizeof(float3)));

    gpuErrchk  ( hipMemcpy(d_viewpoints, h_viewpoints, g_vp_size*sizeof(float3), hipMemcpyHostToDevice));

    g_is_initialized = true;
}

/**
 * Resets the array on the GPU that contains the occlusion results for each test point to all visible.
 * Use after having processed an occlusion query for a given view point and saved its results in order to prepare
 * for a successive query
 */
inline void resetDeviceOutput()
{
    hipMemset(d_outputs, true, g_vertices_size);
}

/**
 * Decallocate memory previously allocated by setGeometryData_interface()
 */
void deleteGeometryData_interface()
{
    free(h_vertex0);
    free(h_vertex1);
    free(h_vertex2);
    free(h_test_points);
    free(h_viewpoints);
    free(h_outputs);
    
    h_vertex0 = NULL;
    h_vertex1 = NULL;
    h_vertex2 = NULL;
    h_test_points = NULL;
    h_viewpoints = NULL;
    h_outputs = NULL;

    hipFree(d_vertex0);
    hipFree(d_vertex1);
    hipFree(d_vertex2);
    hipFree(d_test_points);
    hipFree(d_viewpoints);
    hipFree(d_outputs);

    d_vertex0 = NULL;
    d_vertex1 = NULL;
    d_vertex2 = NULL;
    d_test_points = NULL;
    d_viewpoints = NULL;
    d_outputs = NULL;

    free(h_viewpoints);
    h_viewpoints = NULL;

    hipFree(d_viewpoints);
    d_viewpoints = NULL;

    gpuErrchk(hipDeviceReset());

    g_is_initialized = false;
}

/**
 * CUDA-Kernel for parallelized Moeller-Trumbore ray triangle intersection test. The kernel tests all 
 * mesh triangles for occlusion of a single given test point from a given ray origin.
 * \param ray_origin Pointer to single starting point of the ray for the specific kernel launch
 * \param vertex0_1_2 Pointer to array of vertices of all potential ocludee triangles
 * \param test_point Pointer to the point tested in the specific kernel launch
 * \param output Content of the pointer is set to false if one occlusion is found, left unchanged otherwise
 */
__global__ void occlusionCheck_gpu(float3 *ray_origin, float3 *vertex0, float3 *vertex1, float3 *vertex2, float3 *test_point, bool *output)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    float3 ray_vector = *test_point - *ray_origin;
    ray_vector = (1.0/(norm(ray_vector)))*ray_vector; //Normalize

    float3 edge1, edge2, pvec, tvec, qvec;
    float determinant,inv_det,u,v;

    // find vectors for two edges sharing vertex0
    edge1 = vertex1[i] - vertex0[i];
    edge2 = vertex2[i] - vertex0[i];

    //begin calculating determinant - also used to calculate U parameter
    pvec = cross(ray_vector, edge2);

    determinant = dot(edge1,pvec);
    //TO-DO: Consider including incidence angle check here
    //TO-DO: Consider including back face culling from original paper
    //If determinant is near zero, ray lies in plane of triangle
    if (determinant > -EPSILON && determinant < EPSILON)
    {
        return;
    }

    inv_det = 1.0/determinant;
    tvec = *ray_origin - vertex0[i];
    u = inv_det * dot(tvec,pvec);

    //If any single barycentric is sufficiently close to 1, the respective point itself is tested. If so, skip the triangle in question.
    //If a barycentric is exactly 0, the ray may be tangent to the line connecting the other two barycentric points. If so, continue with checks.
    if (u < 0.0 || u >= 1.0-TOLERANCE)
    {
        return;
    }

    qvec  = cross(tvec, edge1);
    v = inv_det*dot(ray_vector,qvec);
    
    if (v < 0.0 || v >= 1.0-TOLERANCE)
    {
        return;
    }

    //Neccessary because boundary intersection is supposed to pass the test (same vertex for multiple triangles) unlike in original algorithm
    //TO-DO: Check with prior statements
    float w = 1 - u - v;
    if(w <= 0.0 || w >= 1.0-TOLERANCE)
    {
        return;
    }
    double t = inv_det * dot(edge2,qvec);
    if (t > EPSILON) // ray intersects triangle
    {
        //Check whether the point is in front of the triangle
        float distp = norm(*test_point-*ray_origin);
        if(t <= distp)
        {
            *output = false;
        }
    }
}

/**
 * CPU implementation of Moeller Trumbore Ray Triangle Intersection Algorithm
 * \param vp_number Index of the vector of view points passed during previous initialization that is to be checked
 * \returns Boolean vector that states whether the entries vertices vector previously passed are visible
 */
std::vector<int> occlusionCheck_cpu(int vp_number)
{
    float3 ray_origin = h_viewpoints[vp_number];

    std::vector<int> occlusion_res(g_vertices_size, true);
    
    //Looping over all vertices 
    for(int vertex_counter=0; vertex_counter<g_vertices_size; vertex_counter++)
    {
        float3 test_point = h_test_points[vertex_counter];
        bool is_visible = true;
        //Looping over all triangles to see if the vertex under test is occluded
        for(int tri_counter=0; tri_counter<g_mesh_size; tri_counter++)
        {
            float3 vertex0 = h_vertex0[tri_counter];
            float3 vertex1 = h_vertex1[tri_counter];
            float3 vertex2 = h_vertex2[tri_counter];

            float3 ray_vector = test_point - ray_origin;
            ray_vector = (1.0/(norm(ray_vector)))*ray_vector; //Normalize

            float3 edge1, edge2, pvec, tvec, qvec;
            float determinant,inv_det,u,v;
            edge1 = vertex1 - vertex0;
            edge2 = vertex2 - vertex0;

            pvec = cross(ray_vector, edge2);
            determinant = dot(edge1,pvec);
            //TO-DO: Consider including incidence angle check here
            //If determinant is near zero, ray lies in plane of triangle
            if (determinant > -EPSILON && determinant < EPSILON)
            {
                continue;
            }

            inv_det = 1.0/determinant;
            tvec = ray_origin - vertex0;
            u = inv_det * dot(tvec,pvec);

            //If any single barycentric is sufficiently close to 1, the respective point itself is tested. If so, skip the triangle in question.
            //If a barycentric is exactly 0, the ray may be tangent to the line connecting the other two barycentric points. If so, continue with checks.
            if (u < 0.0 || u >= 1.0-TOLERANCE)
            {
                continue;
            }

            qvec  = cross(tvec, edge1);
            v = inv_det*dot(ray_vector,qvec);
            
            if (v < 0.0 || v >= 1.0-TOLERANCE)
            {
                continue;
            }

            //TO-DO: Check with prior statements
            float w = 1 - u - v;
            if(w <= 0.0 || w >= 1.0-TOLERANCE)
            {
                continue;
            }

            double t = inv_det * dot(edge2,qvec);
            if (t > EPSILON) // ray intersection
            {
                //Check whether the point is in front of the triangle
                float distp = norm(test_point-ray_origin);
                if(t <= distp)
                {
                    is_visible = false;
                    break;
                }
            }
        }
        occlusion_res[vertex_counter] = is_visible;
    }
    return occlusion_res;
}

/**
 * Performs an occlusion check from the perspective specified the given view point number using Moeller-Trumbore ray triangle intersection test 
 * (Moeller, Trumbore: "Fast, minimum storage ray-triangle intersection.", Journal of Graphics Tools, 2(1):21--28, 1997. )
 * \param vp_number Index of the vector of view points passed during previous initialization that is to be checked
 * \returns Boolean vector that states whether the entries vertices vector previously passed are visible
 */
std::vector<int> occlusionCheck_interface(int vp_number)
{
    if(!g_is_initialized)
    {
        throw std::runtime_error("No geometry data. Initialization routine needs to be called prior to occlusion check");
    }
    if(vp_number > g_vp_size)
    {
        throw std::runtime_error("vp_number parameter passed exceeds the number view points passed during initialization");
    }

    if(g_use_gpu)
    {
        for(int vertex_counter=0; vertex_counter<g_vertices_size; vertex_counter++)
        {
            occlusionCheck_gpu <<<GRID_SIZE, THREADS_PER_BLOCK>>> (&d_viewpoints[vp_number], 
                                                d_vertex0, 
                                                d_vertex1, 
                                                d_vertex2, 
                                                &d_test_points[vertex_counter], 
                                                &d_outputs[vertex_counter]);

            gpuErrchk( hipPeekAtLastError() );
        }
        // Wait for the GPU to finish
        gpuErrchk  ( hipDeviceSynchronize());
        // Copy result back to host
        gpuErrchk  ( hipMemcpy(h_outputs, d_outputs, g_vertices_size*sizeof(bool), hipMemcpyDeviceToHost));

        //Reset occlusion results in GPU memory to all true
        resetDeviceOutput();

        std::vector<int> output_vect(h_outputs, h_outputs+g_vertices_size);
        return output_vect;
    }
    else
    {
        return occlusionCheck_cpu(vp_number);
    }
}